#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define uint unsigned int

#ifdef __INTELLISENSE__
void __syncthreads();
#endif // __INTELLISENSE__

void InitCPUData(double** matrices, int size);
hipError_t InitGPUData(double** matrices, int **dSize, int size);
hipError_t TransferGPUData(double** matrices, int size, hipMemcpyKind flag);

void ForwardElimination(double* matrix, int size);
void BackwardSubstitute(double* matrix, int size);

__global__ void ForwardEliminationColumn(double* matrix, int* size, int* row);

int main()
{
	int stride = 0;
	dim3 grid_dim = dim3(1, 1, 1);
	dim3 block_dim = dim3(1024, 1, 1);

	int size = 12;										//Number of Rows/Columns, number of elements = size^2 + size
	int *dSize = 0;
	double** matrices = (double**)malloc(3 * sizeof(double*));	//0 CPU, 1 HGPU, 2 DGPU

																//Init matrices and variable storage
	InitCPUData(matrices, size);
	if (InitGPUData(matrices, &dSize, size) != hipSuccess)
	{
		goto Error;
	}
	ForwardElimination(matrices[0], size);
	//KERNEL CALL 1, Forward elimination
	int* dRow = 0;
	hipMalloc((void**)&dRow, sizeof(int));
	for (int i = 1; i < size; ++i)
	{
		hipMemcpy(dRow, &i, sizeof(int), hipMemcpyHostToDevice);
		ForwardEliminationColumn<<<grid_dim, block_dim>>>(matrices[2], dSize, dRow);
	}
	TransferGPUData(matrices, size, hipMemcpyDeviceToHost);
	BackwardSubstitute(matrices[0], size);
	BackwardSubstitute(matrices[1], size);
	bool failed = false;

	for (int i = 0; i < size; ++i)
	{
		if (matrices[0][size] != matrices[1][size])
		{
			failed = true;
			break;
		}
	}
	if(failed)
		printf("Bad result\n");
	else
	{
		for (int i = 1; i < (size + 1); ++i)
			printf("%f\t", matrices[1][i * size + i - 1]);
	}
	printf("\n");
Error:
	free(matrices[0]);
	free(matrices[1]);
	hipFree(matrices[2]);
	free(matrices);
	hipFree(dSize);
	hipFree(dRow);
	system("PAUSE");
	return 0;
}

void InitCPUData(double** matrices, int size)
{
	srand((uint)time(NULL));
	//malloc number of rows
	matrices[0] = (double*)malloc(size * (size + 1) * sizeof(double*));
	matrices[1] = (double*)malloc(size * (size + 1) * sizeof(double*));

	for (int i = 0; i < size; ++i)
	{
		//fill row
		for (int j = 0; j < (size + 1); ++j)
		{
			matrices[0][i * (size + 1) + j] = matrices[1][i * (size + 1) + j] = (double)(rand() % 10 + 1); //not allowing zeros b/c easier
		}
	}
}

hipError_t InitGPUData(double** matrices, int **dSize, int size)
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&matrices[2], size * (size + 1) * sizeof(double*));
	if (cudaStatus == hipSuccess)
	{
		cudaStatus = TransferGPUData(matrices, size, hipMemcpyHostToDevice);
		if (cudaStatus == hipSuccess)
		{
			cudaStatus = hipMalloc((void**)dSize, sizeof(int)); //double void pointer super imoprtant
			if (cudaStatus == hipSuccess)
			{
				cudaStatus = hipMemcpy((void*)*dSize, &size, sizeof(int), hipMemcpyHostToDevice); //maybe move this to TransferGPUData?
				if (cudaStatus != hipSuccess)
				{
					printf("\nCould not copy size variable from host to device\n");
				}
			}
			else
			{
				printf("\nCould not allocate device memory for matrix size\n");
			}
		}
	}
	else
	{
		printf("\nCould not allocate device memory for matrix\n");
	}



	return cudaStatus;
}

hipError_t TransferGPUData(double** matrices, int size, hipMemcpyKind flag)
{
	hipError_t cudaStatus;
	int to = (flag == 1) + 1, from = (flag == 2) + 1;
	cudaStatus = hipMemcpy(matrices[to], matrices[from], size * (size + 1) * sizeof(double), flag);
	if (cudaStatus != hipSuccess)
	{
		printf("\nCould not copy matrix from ");
		flag == 1 ? printf("host to device\n") : printf("device to host\n");
	}
	return cudaStatus;
}

void ForwardElimination(double* matrix, int size)
{
	for (int i = 1; i < size; ++i)
	{
		for (int j = i; j < size; ++j)
		{
			//Calculate ratio between rows, so one can be reduced to 0
			double ratio = matrix[j * (size + 1) + i - 1] / matrix[(i - 1) * (size + 1) + (i - 1)]; //(i - 1) * (size + 2)
			for (int k = 0; k < (size + 1); ++k)
			{
				matrix[j * (size + 1) + k] -= (ratio * matrix[(i - 1) * (size + 1) + k]);
			}
		}
	}
}

void BackwardSubstitute(double* matrix, int size)
{
	for (int i = (size - 1); i > 0; --i)
	{
		matrix[i * (size + 1) + size] = matrix[i * (size + 1) + size] / matrix[i * (size + 1) + i];
		for (int j = i - 1; j > -1; --j)
		{
			//Subtract from the rightmost element
			matrix[j * (size + 1) + size] -= matrix[j * (size + 1) + i] * matrix[i * (size + 1) + size];
			//Eliminate element above
			matrix[j * (size + 1) + i] = 0;
		}
		matrix[i * (size + 1) + i] = 1.f;
	}
	matrix[size] = matrix[size] / matrix[0];
	matrix[0] = 1.f;
}

__global__ void ForwardEliminationColumn(double* matrix, int* size, int* row/*, int* stride*/)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int _size = *size;
	int _row = *row;
	//int _stride = *stride;
	//parallellize over k in ForwardElimination
	if (column < (_size + 1))
	{
		for (int i = _row; i < (_size + 1); ++i)
		{
			//for (int j = 0; j < _stride; ++j)
			//{
			//Calculate ratio between rows, so one can be reduced to 0
			double ratio = (double)matrix[i * (_size + 1) + _row - 1/* + j*/] / (double)matrix[(_row - 1) * (_size + 1) + _row - 1/* + j*/];
			matrix[i * (_size + 1) + column/* + j*/] -= (ratio * matrix[(_row - 1) * (_size + 1) + column/* + j*/]);
			__syncthreads();
			//}
		}
	}
}