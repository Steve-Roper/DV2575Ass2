
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

void InitMatrix(float*** matrix, float** variables, int size);
void Backropagate(float** matrix, int size);
void ForwardSubstitute(float** matrix, float* variablesint, int size);

int main()
{
	float** matrix = 0;
	float* variables = 0;
	int size = 2;
	InitMatrix(&matrix, &variables, size);

Error:
	for (int i = 0; i < size; ++i)
		free(matrix[i]);
	free(matrix);
    return 0;
}

void InitMatrix(float*** matrix, float** variables, int size)
{
	srand(time(NULL));
	//malloc number of rows
	*matrix = (float**)malloc(size * sizeof(float*));
	for (int i = 0; i < size; ++i)
	{
		//malloc a row
		(*matrix)[i] = (float*)malloc((size + 1) * sizeof(float));
		//fill row
		for (int j = 0; j < (size + 1); ++j)
		{
			(*matrix)[i][j] = (float)(rand() % 10 + 1); //not allowing zeros b/c easier
		}
	}
	//malloc variables (x,y,z etc.)
	*variables = (float*)malloc(size * sizeof(float*));
}

void Backropagate(float** matrix, int size)
{
	for (int i = 1; i < size; ++i)
	{
		//Calculate ratio between rows, so one can be reduced to 0
		float ratio = (float)matrix[i][i - 1] / (float)matrix[i - 1][i - 1];
		for (int j = 0; j < size + 1; ++j)
		{
			matrix[i][j] -= (ratio * matrix[i - 1][j]);
		}
	}
}

void ForwardSubstitute(float** matrix, int size, float* variables)
{
	for (int i = (size - 1); i > 0; --i)
	{
		//variables here would usually be x,y,z etc. as in a1x + b1y + c1z = s1
		//												   a2x + b2y + c2z = s2
		//												   a3x + b3y + c3z = s3
		variables[i] = matrix[i][size] / matrix[i][i];
		for (int j = i - 1; j > -1; ++j)
		{
			//Subtract from the rightmost element
			matrix[j][size] -= matrix[j][i] * variables[i];
			//Eliminate element above
			matrix[j][i] = 0;
		}
	}
	variables[0] = matrix[0][size] / matrix[0][0];
}